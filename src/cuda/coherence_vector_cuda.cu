#include "hip/hip_runtime.h"
/**
TODO:
   1. double/Double
   2. Il problema delle celle Fixed/Input è risolto settando tutti i vicini a -1
   3. Parametri generate_next_clock. Valutare possibilità di generare next_clock nel kernel.
   4. Valutare la possibilità di rendere le dimensioni degli array e delle matrici multipli di BLOCK_DIM in modo da eliminare gli "if" nel kernel.
   5. Nel caso in cui double sia sufficiente, ottimizzare letture e scritture con double3
   6. Le define sparse per il codice sono state copiate anzicchè includere gli header... non è il massimo.
*/


#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"

extern "C"{
#include "design.h"
#include "objects/QCADCell.h"
#include "exp_array.h"
#include "coherence_vector.h"
}

#include <math.h>
#include <stdlib.h>
#include <string.h>

#undef	CLAMP
#define	CLAMP(x, low, high)  (((x) > (high)) ? (high) : (((x) < (low)) ? (low) : (x)))
#define	BLOCK_DIM 64
#define	magnitude_energy_vector(P,G) (hypot(2*(G), (P)) * over_hbar) /* (sqrt((4.0*(G)*(G) + (P)*(P))*over_hbar_sqr)) */

// Physical Constants (from coherence_vector.h)
#define hbar 1.05457266e-34
#define over_hbar 9.48252e33
#define hbar_sqr 1.11212e-68
#define over_hbar_sqr 8.99183e67
#define kB 1.381e-23
#define over_kB 7.24296e22
#define E 1.602e-19

// Simulation Types (from global_consts.h)
#define EXHAUSTIVE_VERIFICATION 0
#define VECTOR_TABLE 1

// Simulation Algorithms (from global_consts.h)
#define RUNGE_KUTTA 1
#define EULER_METHOD 2

// Some useful physical constants (from global_consts.h)
#define QCHARGE_SQUAR_OVER_FOUR 6.417423538e-39
#define QCHARGE 1.602176462e-19
#define HALF_QCHARGE 0.801088231e-19
#define OVER_QCHARGE 6.241509745e18
#define ONE_OVER_FOUR_HALF_QCHARGE 3.12109e18
#define EPSILON 8.8541878e-12
#define PI 3.14159265358979323846264338327950288419716939937510582097494459230781640628620899862803482534211706798214808651328230664709384460955058223172535940812848111745028410270193852110555964462294895493038196  
#define FOUR_PI 12.56637061
#define FOUR_PI_EPSILON 1.112650056e-10
#define HBAR 1.0545887e-34
#define PRECISION 1e-5

// Debug-related defines
//#define DEBUG_ON

// Coherence Optimization
__constant__ double optimization_options_clock_prefactor;
__constant__ double optimization_options_clock_shift;
__constant__ double optimization_options_four_pi_over_number_samples;
__constant__ double optimization_options_two_pi_over_number_samples;
__constant__ double optimization_options_hbar_over_kBT;

// Coherence Options
__constant__ double options_clock_low;
__constant__ double options_clock_high;
__constant__ double options_clock_shift;
__constant__ double options_relaxation;
__constant__ double options_time_step;
__constant__ int options_algorithm;

// Other constants
__constant__ double clock_total_shift;


__global__ void kernelIterationParallel 
(
	double *d_polarization, 
	double *d_lambda_x, 
	double *d_lambda_y, 
	double *d_lambda_z, 
	double *d_Ek, 
	unsigned int *d_clock,
	int *d_neighbours, 
	int cells_number, 
	int neighbours_number, 
	int sample_number, 
	int total_number_of_inputs
)
{

   int th_index; 
   int nb_index;   // Neighbour index
   int i;
   double clock_value;
   double PEk;
   double lambda_x, next_lambda_x;
   double lambda_y, next_lambda_y;
   double lambda_z, next_lambda_z;
	double k1, k2, k3, k4;
	double mag;
double arg , dmod;

	th_index =  blockIdx.x * blockDim.x + threadIdx.x;   // Thread index

   // Only useful threads must work
   if (th_index < cells_number)
   {
      PEk = 0;
      for (i = 0; i < neighbours_number; i++)
      {
			if (d_neighbours[th_index*neighbours_number+i] != -1)
			{
	 			nb_index = d_neighbours[th_index*neighbours_number+i];
	 			PEk = PEk + (d_polarization[nb_index] * d_Ek[th_index*neighbours_number+i]) ;
			}
      }




	arg = ((double) (1 << total_number_of_inputs)) * (double)sample_number *  optimization_options_four_pi_over_number_samples - PI * (double)d_clock[th_index] * 0.5;
	
	
	for (dmod = arg; dmod > 0; dmod = dmod - 2*PI);
		
	dmod = dmod + 2*PI;
	if ((dmod < (PI/2+0.00001*PI) && dmod > (PI/2-0.00001*PI)) || (dmod < (3*PI/2+0.00001*PI) && dmod > (3*PI/2-0.00001*PI))) 
			cuPrintf("Cosine function (hazardous arg): arg: %.15g, cos: %.15g\n", arg, cos(arg));



		// Generate clock
		clock_value = optimization_options_clock_prefactor * cos(((double) (1 << total_number_of_inputs)) * (double)sample_number *  optimization_options_four_pi_over_number_samples - PI * (double)d_clock[th_index] * 0.5) + clock_total_shift;
		
		if ( clock_value > options_clock_high )
		{
			clock_value = options_clock_high;
		}
		if ( clock_value < options_clock_low )
		{
			clock_value = options_clock_low;
		}


		// subsequent calls
      lambda_x = d_lambda_x[th_index];
      lambda_y = d_lambda_y[th_index];
      lambda_z = d_lambda_z[th_index];

		
		// LAMBDA_X-----------------------------------------------------------------------

   	mag = magnitude_energy_vector (PEk, clock_value);
   
		k1 = options_time_step * (-(2.0 * clock_value * over_hbar / mag * tanh (optimization_options_hbar_over_kBT * mag) + lambda_x) / options_relaxation + (PEk * lambda_y * over_hbar));

		if (RUNGE_KUTTA == options_algorithm)
		{
		   k2 = options_time_step * (-(2.0 * clock_value * over_hbar / mag * tanh (optimization_options_hbar_over_kBT * mag) + (lambda_x + k1/2)) / options_relaxation + (PEk * lambda_y * over_hbar));
		   k3 = options_time_step * (-(2.0 * clock_value * over_hbar / mag * tanh (optimization_options_hbar_over_kBT * mag) + (lambda_x + k2/2)) / options_relaxation + (PEk * lambda_y * over_hbar));
		   k4 = options_time_step * (-(2.0 * clock_value * over_hbar / mag * tanh (optimization_options_hbar_over_kBT * mag) + (lambda_x + k3)) / options_relaxation + (PEk * lambda_y * over_hbar));
		   next_lambda_x = lambda_x + k1/6 + k2/3 + k3/3 + k4/6;
		}
		else if (EULER_METHOD == options_algorithm)
		   next_lambda_x = lambda_x + k1;
		else
		   next_lambda_x = 0;

		//----------------------------------------------------------------------------------


		// LAMBDA_Y-----------------------------------------------------------------------

   	k1 = options_time_step * -(options_relaxation * (PEk * lambda_x + 2.0 * clock_value * lambda_z) + hbar * lambda_y) / (options_relaxation * hbar);

		if (RUNGE_KUTTA == options_algorithm)
		{
			k2 = options_time_step * -(options_relaxation * (PEk * lambda_x + 2.0 * clock_value * lambda_z) + hbar * (lambda_y + k1/2)) / (options_relaxation * hbar);
			k3 = options_time_step * -(options_relaxation * (PEk * lambda_x + 2.0 * clock_value * lambda_z) + hbar * (lambda_y + k2/2)) / (options_relaxation * hbar);
			k4 = options_time_step * -(options_relaxation * (PEk * lambda_x + 2.0 * clock_value * lambda_z) + hbar * (lambda_y + k3)) / (options_relaxation * hbar);
			next_lambda_y = lambda_y + k1/6 + k2/3 + k3/3 + k4/6;
		}
		else if (EULER_METHOD == options_algorithm)
			next_lambda_y = lambda_y + k1;
		else
			next_lambda_y = 0;

		//--------------------------------------------------------------------------------


		// LAMBDA_Z------------------------------------------------------------------------
		k1 = options_time_step * (PEk * tanh (optimization_options_hbar_over_kBT * mag) + mag * (2.0 * clock_value * options_relaxation * lambda_y - hbar * lambda_z)) / (options_relaxation * hbar * mag);
		if  (optimization_options_hbar_over_kBT*mag)
		if (RUNGE_KUTTA == options_algorithm)
		{
		   k2 = options_time_step * (PEk * tanh (optimization_options_hbar_over_kBT * mag) + mag * (2.0 * clock_value * options_relaxation * lambda_y - hbar * (lambda_z + k1/2))) / (options_relaxation * hbar * mag);
		   k3 = options_time_step * (PEk * tanh (optimization_options_hbar_over_kBT * mag) + mag * (2.0 * clock_value * options_relaxation * lambda_y - hbar * (lambda_z + k2/2))) / (options_relaxation * hbar * mag);
		   k4 = options_time_step * (PEk * tanh (optimization_options_hbar_over_kBT * mag) + mag * (2.0 * clock_value * options_relaxation * lambda_y - hbar * (lambda_z + k3))) / (options_relaxation * hbar * mag);
		   next_lambda_z = lambda_z + k1/6 + k2/3 + k3/3 + k4/6;
		}
		else if (EULER_METHOD == options_algorithm)
		   next_lambda_z = lambda_z + k1;
		else
		   next_lambda_z = 0;

		//-----------------------------------------------------------------------------------------

      d_lambda_x[th_index] = next_lambda_x;
      d_lambda_y[th_index] = next_lambda_y;
      d_lambda_z[th_index] = next_lambda_z;
      
   }
}


/**
 \param <h_polarization> {Vector containing cells polarization (Host side).}
 \param <h_clock> {}
 \param <h_Ek> {}
 \param <h_neighbours> {}
 \param <cells_number> {}
 \param <neighbours_number> {}
 \param <iteration> {}
*/

extern "C"
void launch_coherence_vector_simulation 
(
	DESIGN *design, 
	simulation_data *sim_data, 
	QCADCell ***sorted_cells, 
	coherence_optimizations *optimization_options, 
	const coherence_OP *options, 
	int number_of_cell_layers, 
	int *number_of_cells_in_layer, 
	int num_samples, 
	int record_interval,
	int total_number_of_inputs
)
{
	// Host-Side Variables
	double *h_polarization, *h_Ek, *h_lambda_x, *h_lambda_y, *h_lambda_z;
	unsigned int *h_clock;
	int *h_neighbours;

	// Device-Side Variables
	double *d_polarization, *d_Ek, *d_lambda_x, *d_lambda_y, *d_lambda_z;
	unsigned int *d_clock;
	int *d_neighbours;

	// Others
	#ifdef DEBUG_ON
	FILE *fp;
	#endif
   int i, j, k, l;
   unsigned int cells_number;
   unsigned int max_neighbours_number;
   unsigned int index;
	BUS_LAYOUT_ITER bli ;
  	double dPolarization = 2.0 ;
  	int idxMasterBitOrder = -1.0 ;
   double total_clock_shift = (optimization_options->clock_shift) + options->clock_shift;

	// Compute the number of cells, the max neighbours count and set the cuda_id field of each cell
   cells_number = 0;
   max_neighbours_number = 0;
   index = 0;
   for (i = 0; i < number_of_cell_layers; i++)
   {
      for (j = 0; j < number_of_cells_in_layer[i]; j++)
      {
	 		if (((coherence_model *)sorted_cells[i][j]->cell_model)->number_of_neighbours > max_neighbours_number)
	    		max_neighbours_number = ((coherence_model *)sorted_cells[i][j]->cell_model)->number_of_neighbours;
   
         sorted_cells[i][j]->cuda_id = index;

         index++;
      }
      cells_number += number_of_cells_in_layer[i];
   }

  	// Set GPU Parameters
   dim3 threads (BLOCK_DIM);
   dim3 grid (ceil ((double)cells_number/BLOCK_DIM));

   // Set Devices
   hipSetDevice (cutGetMaxGflopsDeviceId());

   // Allocate CUDA-Compatible Structures
   h_polarization =	(double*) malloc (sizeof(double)*cells_number);
   h_clock =			(unsigned int*) malloc (sizeof(unsigned int)*cells_number);
   h_lambda_x =		(double*) malloc (sizeof(double)*cells_number);
   h_lambda_y =		(double*) malloc (sizeof(double)*cells_number);
   h_lambda_z =		(double*) malloc (sizeof(double)*cells_number);
   h_Ek =				(double*) malloc (sizeof(double)*cells_number*max_neighbours_number);
   h_neighbours =		(int*) malloc (sizeof(int)*cells_number*max_neighbours_number);

	// Fill CUDA-Compatible Structures 
	index = 0;
	#ifdef DEBUG_ON
	fp = fopen ("cuda/log_coherence/circuit_structure", "w");
	#endif
	for (i = 0; i < number_of_cell_layers; i++)
  	{
		for (j = 0; j < number_of_cells_in_layer[i]; j++)
  		{
			h_polarization[index] = qcad_cell_calculate_polarization(sorted_cells[i][j]);
		   h_lambda_x[index] = ((coherence_model *)sorted_cells[i][j]->cell_model)->lambda_x;
		   h_lambda_y[index] = ((coherence_model *)sorted_cells[i][j]->cell_model)->lambda_y;
		   h_lambda_z[index] = ((coherence_model *)sorted_cells[i][j]->cell_model)->lambda_z;
			h_clock[index] = (sorted_cells[i][j]->cell_options).clock;
			#ifdef DEBUG_ON
			fprintf (fp, "Cell: %d, Initial Polarization: %g\n\tNeighbours (Ek):\n", index, h_polarization[index]);
			#endif
			for (k = 0; k < max_neighbours_number; k++)
	   	{
	   		if (k < ((coherence_model *)sorted_cells[i][j]->cell_model)->number_of_neighbours)
      		{
         		h_Ek[index*max_neighbours_number+k] = ((coherence_model *)sorted_cells[i][j]->cell_model)->Ek[k];
         		h_neighbours[index*max_neighbours_number+k] = (((coherence_model *)sorted_cells[i][j]->cell_model)->neighbours[k])->cuda_id;
      		}
      		else
      		{
         		h_Ek[index*max_neighbours_number+k] = -1;
         		h_neighbours[index*max_neighbours_number+k] = -1;
      		}
      		#ifdef DEBUG_ON
				fprintf (fp, "\t\t%d(%g)\n", h_neighbours[index*max_neighbours_number+k], h_Ek[index*max_neighbours_number+k]);
				#endif
	   	}	
			index++;
			#ifdef DEBUG_ON
			fprintf (fp, "\n");
			#endif
		}	
	}
	#ifdef DEBUG_ON
	fclose (fp);
	#endif

   // Initialize Device Memory
   cutilSafeCall (hipMalloc (&d_polarization, cells_number*sizeof(double)));
   cutilSafeCall (hipMalloc (&d_clock, cells_number*sizeof(unsigned int)));
   cutilSafeCall (hipMalloc (&d_lambda_x, cells_number*sizeof(double)));
   cutilSafeCall (hipMalloc (&d_lambda_y, cells_number*sizeof(double)));
   cutilSafeCall (hipMalloc (&d_lambda_z, cells_number*sizeof(double)));
   cutilSafeCall (hipMalloc (&d_Ek, sizeof(double)*max_neighbours_number*cells_number));
   cutilSafeCall (hipMalloc (&d_neighbours, sizeof(unsigned int)*max_neighbours_number*cells_number));

	// Set Device Memory
	cutilSafeCall (hipMemcpy (d_polarization, h_polarization, cells_number*sizeof(double), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_clock, h_clock, cells_number*sizeof(unsigned int), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_lambda_x, h_lambda_x, cells_number*sizeof(double), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_lambda_y, h_lambda_y, cells_number*sizeof(double), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_lambda_z, h_lambda_z, cells_number*sizeof(double), hipMemcpyHostToDevice));			
	cutilSafeCall (hipMemcpy (d_Ek, h_Ek, sizeof(double)*max_neighbours_number*cells_number, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_neighbours, h_neighbours, sizeof(int)*max_neighbours_number*cells_number, hipMemcpyHostToDevice));

	// Set Constants
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_clock_prefactor"), &(optimization_options->clock_prefactor), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_clock_shift"), &(optimization_options->clock_shift), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_four_pi_over_number_samples"), &(optimization_options->four_pi_over_number_samples), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_two_pi_over_number_samples"), &(optimization_options->two_pi_over_number_samples), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_hbar_over_kBT"), &(optimization_options->hbar_over_kBT), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_clock_low"), &(options->clock_low), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_clock_high"), &(options->clock_high), sizeof(double), 0, hipMemcpyHostToDevice));  
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_clock_shift"), &(options->clock_shift), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_relaxation"), &(options->relaxation), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_time_step"), &(options->time_step), sizeof(double), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_algorithm"), &(options->algorithm), sizeof(int), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("clock_total_shift"), &(total_clock_shift), sizeof(double), 0, hipMemcpyHostToDevice));

	// perform the iterations over all samples //
  	for (j = 0; j < num_samples; j++)
   {
   	if (0 == j % 10000)
      {
   	   // Update the progress bar
			printf ("Percentage: %g\n", (float) j / (float) num_samples);
      }

    	//if (EXHAUSTIVE_VERIFICATION == SIMULATION_TYPE)
      	for (idxMasterBitOrder = 0, design_bus_layout_iter_first (design->bus_layout, &bli, QCAD_CELL_INPUT, &i) ; i > -1 ; design_bus_layout_iter_next (&bli, &i), idxMasterBitOrder++)
        	{
				dPolarization = -sin (((double) (1 << idxMasterBitOrder)) * (double) j * optimization_options->four_pi_over_number_samples) > 0 ? 1 : -1;
				index = exp_array_index_1d (design->bus_layout->inputs, BUS_LAYOUT_CELL, i).cell->cuda_id;
				h_polarization[index] = dPolarization;
        		if (0 == j % record_interval)
          		sim_data->trace[i].data[j/record_interval] = dPolarization;
        }
    	/*else
			// DA SISTEMARE...
      	for (design_bus_layout_iter_first (design->bus_layout, &bli, QCAD_CELL_INPUT, &i) ; i > -1 ; design_bus_layout_iter_next (&bli, &i))
        		if (exp_array_index_1d (pvt->inputs, VT_INPUT, i).active_flag)
          	{
					dPolarization = exp_array_index_2d (pvt->vectors, gboolean, (j*pvt->vectors->icUsed) / number_samples, i) ? 1 : -1;
					index = exp_array_index_1d (pvt->inputs, VT_INPUT, i).input.cuda_id;
					h_polarization[index] = dPolarization;
          		if (0 == j % record_interval)
            		sim_data->trace[i].data[j/record_interval] = dPolarization ;
          } */

    	if (0 == j % record_interval)
      {
      	for (design_bus_layout_iter_first (design->bus_layout, &bli, QCAD_CELL_INPUT, &i) ; i > -1 ; design_bus_layout_iter_next (&bli, &i))
			{
				index = exp_array_index_1d (design->bus_layout->inputs, BUS_LAYOUT_CELL, i).cell->cuda_id;
         	sim_data->trace[i].data[j/record_interval] = h_polarization[index];
			}		
		}

		cutilSafeCall (hipMemcpy (d_polarization, h_polarization, cells_number*sizeof(double), hipMemcpyHostToDevice));

		// Launch Kernel

      //printf ("Iteration# %d...", j); 
	cudaPrintfInit();

	kernelIterationParallel<<< grid, threads >>> (d_polarization, d_lambda_x, d_lambda_y, d_lambda_z, d_Ek, d_clock, d_neighbours, cells_number, max_neighbours_number, j, design->bus_layout->inputs->icUsed);

      // Wait Device
      hipDeviceSynchronize ();
		

cudaPrintfDisplay(stdout, true);

 

cudaPrintfEnd();

		//printf("Complete!\n");

		/*char str[256] = "cuda/log_coherence/";
		char num[10];
	
		sprintf (num, "%i", j);
		strcat (str, num);

		fp = fopen(str, "w");
		for( k = 0; k < cells_number; k++)
		{
			fprintf(fp,"cell %d: %f\n", k, h_lambda_z[k]);
		}

		fclose (fp);
		*/

		// Lambda Z is the new polarization
		cutilSafeCall (hipMemcpy (h_lambda_z, d_lambda_z, cells_number*sizeof(double), hipMemcpyDeviceToHost));

		// -- Set the cell polarizations to the lambda_z value -- //
    	for (k = 0; k < number_of_cell_layers; k++)
      	for (l = 0; l < number_of_cells_in_layer[k]; l++)
        	{
        		// don't simulate the input and fixed cells //
        		if (((QCAD_CELL_INPUT == sorted_cells[k][l]->cell_function) || (QCAD_CELL_FIXED == sorted_cells[k][l]->cell_function)))
          		continue;
        		//if (fabs (((coherence_model *)sorted_cells[k][l]->cell_model)->lambda_z) > 1.0)
				if (fabs (h_lambda_z[sorted_cells[k][l]->cuda_id]) > 1.0)
         	{
					printf ("I had to abort the simulation at iteration %d because the polarization = %e was diverging.\nPossible cause is the time step is too large.\nAlternatively, you can decrease the relaxation time to reduce oscillations.\n",j, h_lambda_z[sorted_cells[k][l]->cuda_id]);
         		printf ("time step was set to %e\n", options->time_step);
          		return;
         	}
        		h_polarization[sorted_cells[k][l]->cuda_id] = h_lambda_z[sorted_cells[k][l]->cuda_id];
        	}

		// Collect all the output data from the simulation
    	if (0 == j % record_interval)
      	for (design_bus_layout_iter_first (design->bus_layout, &bli, QCAD_CELL_OUTPUT, &i) ; i > -1 ; design_bus_layout_iter_next (&bli, &i))
        	{
				index = exp_array_index_1d (design->bus_layout->outputs, BUS_LAYOUT_CELL, i).cell->cuda_id;
				sim_data->trace[total_number_of_inputs + i].data[j/record_interval] = h_polarization[index];
			}

	}

	hipFree(d_polarization);
   hipFree(d_clock);
   hipFree(d_lambda_x);
	hipFree(d_lambda_y);
	hipFree(d_lambda_z);
   hipFree(d_Ek);
   hipFree(d_neighbours);  

	return;

}




