#include "hip/hip_runtime.h"
/* ========================================================================== */
/*                                                                            */
/*  CUDA_bistable_iteration.cu                                                */
/*    0- controllare che non vi siano scritte delle porcate da me medesimo*/
/*  1- valutare possibilit� di unrollare il loop sui neighbours               */
/*  (visto che ne stabiliamo il numero di iterazioni a priori)                */
/*  2- il controllo sulle celle fixed crea una bella divergenza... proposte?  */
/*  3- 19maggio: clock_data troooppo grande					*/
/*  --> meglio farsi una memcpy ogni sample di clock_data[4] e d_polarization
	con i nuovi valori di polarizzazione degli input (ancora DA MODIFICARE!)*/

/* ========================================================================== */


#include <cutil_inline.h>
#include <hip/hip_runtime.h>
//#include "cuPrintf.cu"


#include <math.h>

#define BLOCK_DIM 256

  __global__ void bistable_kernel (float* d_polarization, float *d_next_polarization, int *d_cell_clock, float *d_clock_data, float *d_Ek, int *d_neighbours)
  {

   int thr_idx = blockIdx.x * blockDim.x + threadIdx.x;   // Thread index
   int nb_idx;   // Neighbour index
   int q;
   int current_cell_clock;   //could be 0, 1, 2 or 3
   float new_polarization;
   float polarization_math;
   int cells_number, neighbours_number, sample;
  
   // Only useful threads must work
   if (thr_idx < cells_number)
   {

      if (!(d_neighbours[thr_idx * neighbours_number] == -1)) // if thr_idx corresponding cell type is FIXED or INPUT
      {
        polarization_math = 0;
/*        
	for(q = 0; q < neighbours_number; q++)
        {
         nb_idx = d_neighbours[thr_idx * neighbours_number + q];
         polarization_math += d_Ek[thr_idx * neighbours_number + q] * d_polarization[nb_idx];
        }
         */
         //math = math / 2 * gamma
         current_cell_clock  = d_cell_clock[thr_idx];
         polarization_math /= (2.0 * d_clock_data[sample*4 + current_cell_clock]); // ...abbozzo
         
         // -- calculate the new cell polarization -- //
         // if math < 0.05 then math/sqrt(1+math^2) ~= math with error <= 4e-5
         // if math > 100 then math/sqrt(1+math^2) ~= +-1 with error <= 5e-5
            new_polarization =
              (polarization_math        >  1000.0)   ?  1                 :
              (polarization_math        < -1000.0)   ? -1                 :
              (fabs (polarization_math) <     0.001) ?  polarization_math :
                polarization_math / sqrt (1 + polarization_math * polarization_math) ;
         
          //set the new polarization in next_polarization array  
            d_next_polarization[thr_idx] = new_polarization;
          
          
          // -->>> nel caso volessimo considerare la stabilit�...
          // If any cells polarization has changed beyond this threshold
          // then the entire circuit is assumed to have not converged.          
          //  stable = (fabs (new_polarization - old_polarization) <= tolerance) ;                
          //  d_stability[thr_idx] = stable;
        }
      
	else 
         //for FIXED and INPUT type cells polarization remains the same
         d_next_polarization[thr_idx] = d_polarization[thr_idx]; 
      }
      
    }
   
extern "C"
void launch_bistable_simulation(float *h_polarization, float *h_Ek, int *h_cell_clock, float *h_clock_data, int *h_neighbours, int cells_number, int neighbours_number, int number_of_samples, int iterations_per_sample)
{

printf("\nentrato nella launch!\n");
 // Variables
   float *d_next_polarization, *d_polarization, *d_clock_data, *d_Ek;
   int *d_neighbours, *d_cell_clock;
   int i;

   // Set GPU Parameters
   dim3 threads (BLOCK_DIM);
   dim3 grid (ceil ((float)cells_number/BLOCK_DIM));

   // Set Devices
   hipSetDevice (cutGetMaxGflopsDeviceId());
//   cudaPrintfInit ();

   // Initialize Memory
   cutilSafeCall (hipMalloc (&d_next_polarization, cells_number * sizeof(float))); 
   cutilSafeCall (hipMalloc (&d_polarization, cells_number * sizeof(float))); 
   cutilSafeCall (hipMalloc (&d_cell_clock, cells_number * sizeof(int)));
   cutilSafeCall (hipMalloc (&d_clock_data, 4 * number_of_samples * sizeof(float)));
   cutilSafeCall (hipMalloc (&d_Ek, sizeof(float)*neighbours_number*cells_number));
   cutilSafeCall (hipMalloc (&d_neighbours, sizeof(int)*neighbours_number*cells_number));

printf("malloc eseguite\n");
   // Set Memory
   cutilSafeCall (hipMemcpy (d_polarization, h_polarization, cells_number * sizeof(float), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_cell_clock, h_cell_clock, cells_number * sizeof(int), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_clock_data, h_clock_data, 4 * number_of_samples * sizeof(float), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_Ek, h_Ek, sizeof(float) * neighbours_number * cells_number, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_neighbours, h_neighbours, sizeof(int) * neighbours_number * cells_number, hipMemcpyHostToDevice));
printf("memcpy eseguite\n");

int j;

 for (j = 0; j < number_of_samples ; j++)
  {


  // In each sample...
   for (i = 0; i < iterations_per_sample; i++) //we are not considering stability
   {
      // Launch Kernel
      bistable_kernel<<< grid, threads >>> (d_polarization, d_next_polarization, d_cell_clock, d_clock_data, d_Ek, d_neighbours);

      // Wait Device
    //  hipDeviceSynchronize ();

//printf("j:%d \t i=%d\n", j,i);      

      // Set Memory for the next iteration
   //   cutilSafeCall (hipMemcpy (d_polarization, d_next_polarization, cells_number * sizeof(float), hipMemcpyDeviceToDevice));
      
    }
	// Get desidered iteration results from GPU
 //  cutilSafeCall (hipMemcpy (h_polarization, d_polarization, cells_number * sizeof(float), hipMemcpyDeviceToHost));
  }
      
      printf("\nfacciamo la clean\n");      
// Free-up resources
//   cudaPrintfEnd();
   hipFree(d_next_polarization);
   hipFree(d_polarization);
   hipFree(d_cell_clock);
   hipFree(d_clock_data);
   hipFree(d_Ek);
   hipFree(d_neighbours);  




}
