#include "hip/hip_runtime.h"
/* ========================================================================== */
/*                                                                            */
/*  CUDA_bistable_iteration.cu                                                */
/*    0- controllare che non vi siano scritte delle porcate da me medesimo*/
/*  1- valutare possibilit� di unrollare il loop sui neighbours               */
/*  (visto che ne stabiliamo il numero di iterazioni a priori)                */
/*  2- il controllo sulle celle fixed crea una bella divergenza... proposte?  */
/*  3- 19maggio: clock_data troooppo grande
/*  --> meglio farsi una memcpy ogni sample di clock_data[4] e d_polarization
	con i nuovi valori di polarizzazione degli input (ancora DA MODIFICARE!)
					*/
/* ========================================================================== */
#define CUPRINTF_B

#include <cutil_inline.h>
#include <hip/hip_runtime.h>

#ifdef CUPRINTF_B
#include "cuPrintf.cu"
#endif //CUPRINTF_B

#include <time.h>
extern "C"{
#include "../coloring/coloring.h"
}
#include <math.h>

#define BLOCK_DIM 256
#undef CLAMP
#define CLAMP(value,low,high) ((value > high) ? high : ((value < low) ? low : value))
#undef PI
#define PI  3.14159265358979323846
#undef FOUR_PI
#define FOUR_PI 12.56637061

__device__ __constant__ double d_clock_prefactor;
__device__ __constant__ double d_clock_shift;
__device__ __constant__ int d_cells_number;
__device__ __constant__ int d_neighbours_number;
__device__ __constant__ int d_input_number;
__device__ __constant__ int d_output_number;
__device__ __constant__ int d_number_of_samples;
__device__ __constant__ double d_clock_low;
__device__ __constant__ double d_clock_high;

extern	__shared__ int shm_array[];

__device__ inline int find(int x, int *array, int length)
{
	int l = 0, r = length - 1, mid;
	while (l <= r)
	{
		mid = (l + r) / 2;
		if (x==10191) cuPrintf("is %d?\n",array[mid]);
		if (array[mid] == x) return mid;
		else if (array[mid] > x) r = mid - 1;
		else l = mid + 1;
	}
	return -1;
}


__global__ void update_inputs (double *d_polarization, int *d_input_indexes, int sample)
{
	int input_idx;
    double tmp;
	int thr_idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (threadIdx.x < d_input_number)
	{
		shm_array[threadIdx.x] = d_input_indexes[threadIdx.x];
	}
	__syncthreads();
	
	cuPrintf("%d: ECCOLO: %d\n",thr_idx, shm_array[4]);
	input_idx = find(thr_idx, shm_array, d_input_number);
	cuPrintf("%d: RIECCOLO: %d\n",thr_idx, shm_array[4]);
    //cuPrintf("input idx: %i, input_number: %i sample: %i\n",input_idx,d_input_number,sample);
	if (input_idx >= 0)
	{
		tmp = ((double)( 1 << input_idx)) * (double)sample * 4.0 * PI /(double) d_number_of_samples;
		//cuPrintf("tmp: %e, ",tmp);
		tmp = -1 * sin(tmp);
		//cuPrintf("tmp: %e, ",tmp);
		d_polarization[thr_idx]=(tmp > 0) ? 1: -1;
		//cuPrintf("Ciao sono l'input %d: %e. Input index:",thr_idx,d_polarization[thr_idx]);
		//int i;
		//for (i=0;i<d_input_number;i++) cuPrintf("%d ", shm_array[i]);
		//cuPrintf("\n");
		/*double sin0=sin(0.0);
		double sinf0=__sinf(0.0);
		double cospi2=cos(PI/2);
		double cosfpi2=cosf(PI/2);
		double flsin0=sin(PI/5);
		double flsinf0=__sinf(PI/5);
		cuPrintf("input: %e, sin(0)=%e, __sinf(0)=%e, cos(pi/4)=%e, __cosf(pi/4)=%e, sin(pi/5)=%e, __sinf(pi/5)=%e\n",d_polarization[thr_idx],sin0,sinf0,cospi2,cosfpi2,flsin0,flsinf0);*/
	}
}


__global__ void bistable_kernel (
		double *d_polarization,
		double *d_next_polarization,
		int *d_cell_clock,
		double *d_Ek,
		int *d_neighbours,
		int sample,
		int *d_output_indexes,
		int *d_stability,
		double tolerance,
		double *d_output_data,
		int *d_cells_colors,
		int color
		)
{
	int thr_idx = blockIdx.x * blockDim.x + threadIdx.x;   // Thread index
	int nb_idx;   // Neighbour index
	int q;
	int current_cell_clock;   //could be 0, 1, 2 or 3
	double new_polarization;
	double polarization_math;
	double clock_value;
	int input_idx;
	int output_idx;
	int stable;
	int *shm_output_indexes = shm_array;
	double nb_pol;
	double kink;
	
	if (threadIdx.x < d_output_number)
	{
		shm_output_indexes[threadIdx.x] = d_output_indexes[threadIdx.x];
	}

	__syncthreads();

	// Only useful threads must work
	if (thr_idx < d_cells_number)
	{		
		//cuPrintf("GO! my_color:%d\n",color);
		//cuPrintf("\nd_output_number = %d,\t d_output_indexes[0]=%d\n",d_output_number, d_output_indexes[0] );	
		  
		//cuPrintf("%f ", d_polarization[thr_idx]);	

		if (!(d_neighbours[thr_idx] == -1) && color == d_cells_colors[thr_idx]) // if thr_idx corresponding cell type is not FIXED or INPUT and is my turn
		{
			nb_idx = 0;
			polarization_math = 0;
			for(q = 0; q < d_neighbours_number & nb_idx != -1; q++)
			{
				nb_idx = d_neighbours[thr_idx + q * d_cells_number];
				if (nb_idx != -1) 
				{
					kink = d_Ek[thr_idx + q*d_cells_number];
					polarization_math += kink * d_polarization[nb_idx];
				}
			}

			//math = math / 2 * gamma
			current_cell_clock  = d_cell_clock[thr_idx];
			clock_value = d_clock_prefactor * cos (((double)(1 << d_input_number)) * (double)sample * 4.0 * PI / (double)d_number_of_samples - PI * current_cell_clock / 2) + d_clock_shift;
			clock_value = CLAMP(clock_value,d_clock_low,d_clock_high);
			polarization_math /= (2.0 * clock_value);
			 
			// -- calculate the new cell polarization -- //
			// if math < 0.05 then math/sqrt(1+math^2) ~= math with error <= 4e-5
			// if math > 100 then math/sqrt(1+math^2) ~= +-1 with error <= 5e-5
			new_polarization =
			(polarization_math        >  1000.0)   ?  1                 :
			(polarization_math        < -1000.0)   ? -1                 :
			(fabs (polarization_math) <     0.001) ?  polarization_math :
			polarization_math / sqrt (1 + polarization_math * polarization_math) ;
			
			//set the new polarization in next_polarization array  
			d_next_polarization[thr_idx] = new_polarization;

			// If any cells polarization has changed beyond this threshold
			// then the entire circuit is assumed to have not converged.      
			stable = (fabs (new_polarization - d_polarization[thr_idx]) <= tolerance);
			d_stability[thr_idx] = stable;

			output_idx = find(thr_idx, shm_output_indexes, d_output_number);

			if (output_idx >= 0)
			{
				d_output_data[output_idx] = new_polarization;
			}
		}
		else
		{
			d_next_polarization[thr_idx] = d_polarization[thr_idx];
		}
	}
}

__host__ void swap_arrays(double **array_1, double **array_2)
{
	double *temp = *array_1;
	*array_1 = *array_2;
	*array_2 = temp;
}
   
extern "C"
void launch_bistable_simulation(
	double *h_polarization,
	double *h_Ek,
	int *h_cell_clock,
	int *h_neighbours,
	int cells_number,
	int neighbours_number,
	int number_of_samples,
	int max_iterations,
	int *input_indexes,
	int input_number,
	int *output_indexes,
	int output_number,
	double clock_prefactor,
	double clock_shift,
	double clock_low,
	double clock_high, 
	double tolerance,
	double ***output_traces
	)
{


	// Variables
	double *d_next_polarization, *d_polarization, *d_Ek;
	int *d_neighbours, *d_cell_clock, *d_input_indexes, *d_output_indexes;
	int i,j,stable,color, num_colors;
	int *d_stability, *h_stability, *h_cells_colors, *d_cells_colors;
	int count;
	int k;
	double *d_output_data;
	double *h_output_data;


	
	/*printf("\ntesting launch parameters:\n cells_number = %d\n neighbours_number = %d \n number_of_samples = %d\n max_iterations = %d\n, tolerance = %e\npref: %e, shift: %e, low: %e, high: %e\n",cells_number, neighbours_number, number_of_samples, max_iterations, tolerance,clock_prefactor,clock_shift,clock_low,clock_high);
	printf("output_number = %d, output_indexes[0]= %d\n", output_number , output_indexes[0]);*/


	h_output_data = (double *) malloc(sizeof(double) * output_number);
	h_stability = (int *)malloc(sizeof(int)*cells_number);
	
	//coloring
	color_graph(h_neighbours, cells_number, neighbours_number, &h_cells_colors, &num_colors);
	//debug
	/*printf("Number of samples:%d\nNumber of colors:%d\nColors:\n",number_of_samples, num_colors);
	for (i=0;i<cells_number;i++) printf("%d ",h_cells_colors[i]);
	printf("\n");*/
	
	// Set GPU Parameters
	

	dim3 threads (BLOCK_DIM);
	dim3 grid (ceil ((double)cells_number/BLOCK_DIM));

	// Set Devices
	//hipSetDevice (cutGetMaxGflopsDeviceId());

#ifdef CUPRINTF_B
	cudaPrintfInit ();
#endif

	//starting timer
	timespec startTime, endTime;
	clock_gettime(CLOCK_REALTIME, &startTime);

	
	// Initialize Memory
	cutilSafeCall (hipMalloc ((void**)&d_output_data, output_number * sizeof(double)));
	cutilSafeCall (hipMalloc ((void**)&d_next_polarization, cells_number * sizeof(double)));
	cutilSafeCall (hipMalloc ((void**)&d_polarization, cells_number * sizeof(double))); 
	cutilSafeCall (hipMalloc ((void**)&d_Ek, sizeof(double)*neighbours_number*cells_number));
	cutilSafeCall (hipMalloc ((void**)&d_cell_clock, cells_number * sizeof(int)));
	cutilSafeCall (hipMalloc ((void**)&d_neighbours, sizeof(int)*neighbours_number*cells_number));
	cutilSafeCall (hipMalloc ((void**)&d_input_indexes, sizeof(int)*input_number));
	cutilSafeCall (hipMalloc ((void**)&d_output_indexes, sizeof(int)*output_number));
	cutilSafeCall (hipMalloc ((void**)&d_stability, sizeof(int)*cells_number));
	cutilSafeCall (hipMalloc ((void**)&d_cells_colors, sizeof(int)*cells_number));
	

	// Set Memory

	cutilSafeCall (hipMemcpy (d_next_polarization, h_polarization, cells_number * sizeof(double), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_polarization, h_polarization, cells_number * sizeof(double), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_Ek, (double *)h_Ek, sizeof(double) * neighbours_number * cells_number, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_cell_clock, h_cell_clock, cells_number * sizeof(int), hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_neighbours, h_neighbours, sizeof(int) * neighbours_number * cells_number, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_input_indexes, input_indexes, sizeof(int)*input_number, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_output_indexes, output_indexes, sizeof(int)*output_number, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpy (d_cells_colors, h_cells_colors, sizeof(int)*cells_number, hipMemcpyHostToDevice));
	
	printf("\nIo host mando questi al device:\n");
	for (i=0;i<input_number;i++) printf("%d ", input_indexes[i]);
	printf("\n\n");

	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_clock_prefactor"), &(clock_prefactor), sizeof(double), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_clock_shift"), &(clock_shift), sizeof(double), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_cells_number"), &(cells_number), sizeof(int), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_neighbours_number"), &(neighbours_number), sizeof(int), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_input_number"), &(input_number), sizeof(int), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_output_number"), &(output_number), sizeof(int), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_number_of_samples"), &(number_of_samples), sizeof(double), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_clock_low"), &(clock_low), sizeof(double), 0, hipMemcpyHostToDevice));
	cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("d_clock_high"), &(clock_high), sizeof(double), 0, hipMemcpyHostToDevice));
	
	//srand(time(0));


	for (j = 0; j < 1/*number_of_samples*/; j++)
	{

		stable = 0;

		update_inputs<<< grid, threads >>> (d_polarization, d_input_indexes, j);
		hipDeviceSynchronize ();
		
		
	
		// In each sample...
		for (i = 0; i < 2/*max_iterations && !stable*/; i++)
		{
				
			// Launch Kernel
			for(color = 1; color <= num_colors; color++)
			{
				/*cutilSafeCall(hipMemcpy(h_polarization,d_polarization,cells_number*sizeof(double),hipMemcpyDeviceToHost));
				for (k=0;k<cells_number;k++) printf("i:%d, col:%d, cell:%d\t%e\n",i,color,k,h_polarization[k]);*/
				
				bistable_kernel<<< grid, threads >>> (d_polarization, d_next_polarization, d_cell_clock, d_Ek, d_neighbours, 
					j, d_output_indexes, d_stability, tolerance, d_output_data, d_cells_colors, color);
					
				// Wait Device
				hipDeviceSynchronize ();
				
				// Set Memory for the next iteration
				//			cutilSafeCall (hipMemcpy (d_polarization, d_next_polarization, cells_number * sizeof(double), hipMemcpyDeviceToDevice));
				swap_arrays(&d_polarization,&d_next_polarization);
			}
			//	for (count = 0; count<cells_number; count++) printf("%d",h_stability[count]);
			//	printf("\n");

			
			
			cutilSafeCall (hipMemcpy (h_stability, d_stability, cells_number*sizeof(int), hipMemcpyDeviceToHost));

			count = 0;
			stable = 1;
			while (count<cells_number && h_stability[count] != 0) count++;
			if (count < cells_number) stable = 0;

	//	  	printf("stabilit�: %d,max_iter: %d",stable,max_iterations);

			/*cutilSafeCall (hipMemcpy (h_polarization, d_polarization, cells_number*sizeof(double), hipMemcpyDeviceToHost));
			for (count=0; count<20; count++) printf("%e\t",h_polarization[count]);
			printf("\n");*/


			
		}

		// Get desidered iteration results from GPU
		cutilSafeCall (hipMemcpy (h_output_data, d_output_data, output_number * sizeof(double), hipMemcpyDeviceToHost));

		for (k=0;k<output_number;k++)
		{
			//printf("%e\n", h_output_data[k]); //maybe %lf now that we use double
			(*output_traces)[k][j] = h_output_data[k];
		}

	
		if(j%100 == 0) fprintf(stderr,"#Simulating: %d % \titerations: %d\n", (j*100/number_of_samples), i);

		

	}

#ifdef CUPRINTF_B
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
#endif //CUPRINTF_B



	// Free-up resources
	hipFree(d_output_data);
	hipFree(d_next_polarization);
	hipFree(d_input_indexes);
	hipFree(d_output_indexes);
	hipFree(d_polarization);
	hipFree(d_cell_clock);
	hipFree(d_stability);
	hipFree(d_Ek);
	hipFree(d_neighbours);


	//get time result	
	clock_gettime(CLOCK_REALTIME, &endTime);
		timespec temp;
	if ((endTime.tv_nsec-startTime.tv_nsec)<0)
	{
		temp.tv_sec = endTime.tv_sec-startTime.tv_sec-1;
		temp.tv_nsec = 1000000000+endTime.tv_nsec-startTime.tv_nsec;
	} 
	else
	{
		temp.tv_sec = endTime.tv_sec-startTime.tv_sec;
		temp.tv_nsec = endTime.tv_nsec-startTime.tv_nsec;
	}

	fprintf(stdout, "\tProcessing time1: %f (s)\n", (double)temp.tv_sec);

	fprintf(stdout, "\tProcessing time2: %f (ns)\n", (double)temp.tv_nsec);
}
#undef CUPRINTF_B
