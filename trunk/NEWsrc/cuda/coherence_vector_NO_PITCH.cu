#include "hip/hip_runtime.h"
/**
TODO:
   1. Float/Double
   2. Il problema delle celle Fixed/Input è risolto settando tutti i vicini a -1
   3. Parametri generate_next_clock. Valutare possibilità di generare next_clock nel kernel.
   4. Valutare la possibilità di rendere le dimensioni degli array e delle matrici multipli di BLOCK_DIM in modo da eliminare gli "if" nel kernel.
   5. Nel casdo in cui Float sia sufficiente, ottimizzare letture e scritture con Float3
   6. Le define sparse per il codice sono state copiate anzicchè includere gli header... non è il massimo.
*/


#include <cutil_inline.h>
#include <hip/hip_runtime.h>
#include "cuPrintf.cu"
#include "structs.h"

#include <math.h>

#undef	CLAMP
#define	CLAMP(x, low, high)  (((x) > (high)) ? (high) : (((x) < (low)) ? (low) : (x)))
#define	BLOCK_DIM 256
#define	magnitude_energy_vector(P,G) (hypot(2*(G), (P)) * over_hbar) /* (sqrt((4.0*(G)*(G) + (P)*(P))*over_hbar_sqr)) */

// Physical Constants (from coherence_vector.h)
#define hbar 1.05457266e-34
#define over_hbar 9.48252e33
#define hbar_sqr 1.11212e-68
#define over_hbar_sqr 8.99183e67
#define kB 1.381e-23
#define over_kB 7.24296e22
#define E 1.602e-19

// Simulation Types (from global_consts.h)
#define EXHAUSTIVE_VERIFICATION 0
#define VECTOR_TABLE 1

// Simulation Algorithms (from global_consts.h)
#define RUNGE_KUTTA 1
#define EULER_METHOD 2

// Some useful physical constants (from global_consts.h)
#define QCHARGE_SQUAR_OVER_FOUR 6.417423538e-39
#define QCHARGE 1.602176462e-19
#define HALF_QCHARGE 0.801088231e-19
#define OVER_QCHARGE 6.241509745e18
#define ONE_OVER_FOUR_HALF_QCHARGE 3.12109e18
#define EPSILON 8.8541878e-12
#define PI 3.1415926535897932384626433832795
#define FOUR_PI 12.56637061
#define FOUR_PI_EPSILON 1.112650056e-10
#define HBAR 1.0545887e-34
#define PRECISION 1e-5


// Coherence Optimization
__constant__ float optimization_options_clock_prefactor;
__constant__ float optimization_options_clock_shift;
__constant__ float optimization_options_four_pi_over_number_samples;
__constant__ float optimization_options_two_pi_over_number_samples;
__constant__ float optimization_options_hbar_over_kBT;

// Coherence Options
__constant__ float options_clock_low;
__constant__ float options_clock_high;
__constant__ float options_clock_shift;
__constant__ float options_relaxation;
__constant__ float options_time_step;
__constant__ int options_algorithm;


__device__ inline float slope_x (float t, float PEk, float Gamma, float lambda_x, float lambda_y, float lambda_z)
{
   float mag = magnitude_energy_vector (PEk, Gamma);
   return (-(2.0 * Gamma * over_hbar / mag * tanh (optimization_options_hbar_over_kBT * mag) + lambda_x) / options_relaxation + (PEk * lambda_y * over_hbar));
}


__device__ inline float slope_y (float t, float PEk, float Gamma, float lambda_x, float lambda_y, float lambda_z)
{
   return -(options_relaxation * (PEk * lambda_x + 2.0 * Gamma * lambda_z) + hbar * lambda_y) / (options_relaxation * hbar);
}


__device__ inline float slope_z (float t, float PEk, float Gamma, float lambda_x, float lambda_y, float lambda_z)
{
   float mag = magnitude_energy_vector (PEk, Gamma);
   return (PEk * tanh (optimization_options_hbar_over_kBT * mag) + mag * (2.0 * Gamma * options_relaxation * lambda_y - hbar * lambda_z)) / (options_relaxation * hbar * mag);
}


// Next value of lambda x with choice of options_algorithm
__device__ inline float eval_next_lambda_x (float t, float PEk, float Gamma, float lambda_x, float lambda_y, float lambda_z)
{
   float k1 = options_time_step * slope_x (t, PEk, Gamma, lambda_x, lambda_y, lambda_z);
   float k2, k3, k4;

   if (RUNGE_KUTTA == options_algorithm)
   {
      k2 = options_time_step * slope_x (t, PEk, Gamma, lambda_x + k1/2, lambda_y, lambda_z);
      k3 = options_time_step * slope_x (t, PEk, Gamma, lambda_x + k2/2, lambda_y, lambda_z);
      k4 = options_time_step * slope_x (t, PEk, Gamma, lambda_x + k3,   lambda_y, lambda_z);
      return lambda_x + k1/6 + k2/3 + k3/3 + k4/6;
   }
   else
   if (EULER_METHOD == options_algorithm)
      return lambda_x + k1;
   else
      return 0;
}


// Next value of lambda y with choice of options_algorithm
__device__ inline float eval_next_lambda_y (float t, float PEk, float Gamma, float lambda_x, float lambda_y, float lambda_z)
{
   float k1 = options_time_step * slope_y (t, PEk, Gamma, lambda_x, lambda_y, lambda_z);
   float k2, k3, k4;

   if (RUNGE_KUTTA == options_algorithm)
   {
      k2 = options_time_step * slope_y (t, PEk, Gamma, lambda_x, lambda_y + k1/2, lambda_z);
      k3 = options_time_step * slope_y (t, PEk, Gamma, lambda_x, lambda_y + k2/2, lambda_z);
      k4 = options_time_step * slope_y (t, PEk, Gamma, lambda_x, lambda_y + k3,   lambda_z);
      return lambda_y + k1/6 + k2/3 + k3/3 + k4/6;
   }
   else
   if (EULER_METHOD == options_algorithm)
      return lambda_y + k1;
   else
      return 0;
}


// Next value of lambda z with choice of options_algorithm
__device__ inline float eval_next_lambda_z (float t, float PEk, float Gamma, float lambda_x, float lambda_y, float lambda_z)
{
   float k1 = options_time_step * slope_z (t, PEk, Gamma, lambda_x, lambda_y, lambda_z);
   float k2, k3, k4;

   if (RUNGE_KUTTA == options_algorithm)
   {
      k2 = options_time_step * slope_z(t, PEk, Gamma, lambda_x, lambda_y, lambda_z + k1/2);
      k3 = options_time_step * slope_z(t, PEk, Gamma, lambda_x, lambda_y, lambda_z + k2/2);
      k4 = options_time_step * slope_z(t, PEk, Gamma, lambda_x, lambda_y, lambda_z + k3  );
      return lambda_z + k1/6 + k2/3 + k3/3 + k4/6;
   }
   else
   if (EULER_METHOD == options_algorithm)
      return lambda_z + k1;
   else
      return 0;
}


__device__ inline float generate_clock_at_sample_s (unsigned int clock_num, unsigned long int sample, unsigned long int number_samples, int total_number_of_inputs, const CUDA_coherence_OP *options, int SIMULATION_TYPE /*, VectorTable *pvt*/)
{
/*
   float clock = optimization_options_clock_prefactor * cos (((float) (1 << total_number_of_inputs)) * (float) sample * optimization_options_four_pi_over_number_samples - PI * (float)clock_num * 0.5) + optimization_options_clock_shift + options_clock_shift;

   // Saturate the clock at the clock high and low values
	clock = CLAMP (clock, options_clock_low, options_clock_high) ;
	return clock;
*/
	return CLAMP (optimization_options_clock_prefactor * cos (((float) (1 << total_number_of_inputs)) * (float) sample * optimization_options_four_pi_over_number_samples - PI * (float)clock_num * 0.5) + optimization_options_clock_shift + options_clock_shift, options_clock_low, options_clock_high);
}

/*
__global__ void kernelIterationParallelOptimized (float* d_next_polarization, float *d_polarization, float *d_lambda_x, float *d_lambda_y, float *d_lambda_z, float *d_Ek, int *d_neighbours, int cells_number, int neighbours_number, int sample_number, int number_of_inputs)
{

   int th_index = blockIdx.x * blockDim.x + threadIdx.x;   // Thread index
   int nb_index;   // Neighbour index
   int i;
   float clock_value;
   float PEk;
   float lambda_x, next_lambda_x;
   float lambda_y, next_lambda_y;
   float lambda_z, next_lambda_z;
   float t;
   int total_number_of_inputs = number_of_inputs;

   // Only usefull threads must work
   if (th_index < cells_number)
   {
      t = options_time_step * sample_number;

      // Generate clock
		clock_value = CLAMP (optimization_options_clock_prefactor * cos (((float) (1 << total_number_of_inputs)) * (float) sample_number * optimization_options_four_pi_over_number_samples - PI * (float)clock_num * 0.5) + optimization_options_clock_shift + options_clock_shift, options_clock_low, options_clock_high);

      PEk = 0;
   
      for (i = 0; i < neighbours_number; i++)
      {
	 nb_index = d_neighbours[th_index*neighbours_number+i];
	 PEk += d_polarization[nb_index] * d_Ek[th_index*neighbours_number+nb_index]; 
      }

      lambda_x = d_lambda_x[th_index];
      lambda_y = d_lambda_y[th_index];
      lambda_z = d_lambda_z[th_index];

      next_lambda_x = eval_next_lambda_x (t, PEk, clock_value, lambda_x, lambda_y, lambda_z);
      next_lambda_y = eval_next_lambda_y (t, PEk, clock_value, lambda_x, lambda_y, lambda_z);
      next_lambda_z = eval_next_lambda_z (t, PEk, clock_value, lambda_x, lambda_y, lambda_z);

      d_lambda_x[th_index] = next_lambda_x;
      d_lambda_y[th_index] = next_lambda_y;
      d_lambda_z[th_index] = next_lambda_z;
      
      d_next_polarization[th_index] = next_lambda_z;

      cuPrintf("polarization: %f\tclock: %f\tlambda: %f %f %f\tEk: %f\n", d_polarization[th_index], clock_value, d_lambda_x[th_index], d_lambda_y[th_index], d_lambda_z[th_index], d_Ek[th_index]);
   }

}
*/

__global__ void kernelIterationParallel (float* d_next_polarization, float *d_polarization, float *d_lambda_x, float *d_lambda_y, float *d_lambda_z, float *d_Ek, int *d_neighbours, int cells_number, int neighbours_number, int sample_number, int number_of_inputs)
{

   int th_index = blockIdx.x * blockDim.x + threadIdx.x;   // Thread index
   int nb_index;   // Neighbour index
   int i;
   float clock_value;
   float PEk;
   float lambda_x, next_lambda_x;
   float lambda_y, next_lambda_y;
   float lambda_z, next_lambda_z;
   float t;
   int total_number_of_inputs = number_of_inputs;

   // Only usefull threads must work
   if (th_index < cells_number)
   {
      t = options_time_step * sample_number;

      // Generate clock
		clock_value = 5; //generate_clock_at_sample_s (h_clock, cells_number, i, ...)

      PEk = 0;
   
      for (i = 0; i < neighbours_number; i++)
      {
	 nb_index = d_neighbours[th_index*neighbours_number+i];
	 PEk += d_polarization[nb_index] * d_Ek[th_index*neighbours_number+nb_index]; 
      }

      lambda_x = d_lambda_x[th_index];
      lambda_y = d_lambda_y[th_index];
      lambda_z = d_lambda_z[th_index];

      next_lambda_x = eval_next_lambda_x (t, PEk, clock_value, lambda_x, lambda_y, lambda_z);
      next_lambda_y = eval_next_lambda_y (t, PEk, clock_value, lambda_x, lambda_y, lambda_z);
      next_lambda_z = eval_next_lambda_z (t, PEk, clock_value, lambda_x, lambda_y, lambda_z);

      d_lambda_x[th_index] = next_lambda_x;
      d_lambda_y[th_index] = next_lambda_y;
      d_lambda_z[th_index] = next_lambda_z;
      
      d_next_polarization[th_index] = next_lambda_z;

      cuPrintf("polarization: %f\tclock: %f\tlambda: %f %f %f\tEk: %f\n", d_polarization[th_index], clock_value, d_lambda_x[th_index], d_lambda_y[th_index], d_lambda_z[th_index], d_Ek[th_index]);
   }

}


/**
 \param <h_polarization> {Vector containing cells polarization (Host side).}
 \param <h_clock> {}
 \param <h_Ek> {}
 \param <h_neighbours> {}
 \param <cells_number> {}
 \param <neighbours_number> {}
 \param <iteration> {}
*/
extern "C"
void launch_coherence_vector_simulation (float *h_polarization, float *h_lambda_x, float *h_lambda_y, float *h_lambda_z, float *h_Ek, int *h_neighbours, int cells_number, int neighbours_number, int iterations, CUDA_coherence_OP *options, CUDA_coherence_optimizations *optimization_options)
{

   // Variables
   float *d_next_polarization, *d_polarization, *d_Ek, *d_lambda_x, *d_lambda_y, *d_lambda_z;
   int *d_neighbours;
   int i;

   // Set GPU Parameters
   dim3 threads (BLOCK_DIM);
   dim3 grid (ceil ((float)cells_number/BLOCK_DIM));

   // Set Devices
   hipSetDevice (cutGetMaxGflopsDeviceId());
   cudaPrintfInit ();

   // Initialize Memory
   cutilSafeCall (hipMalloc (&d_next_polarization, cells_number*sizeof(float))); 
   cutilSafeCall (hipMalloc (&d_polarization, cells_number*sizeof(float))); 
   cutilSafeCall (hipMalloc (&d_lambda_x, cells_number*sizeof(float)));
   cutilSafeCall (hipMalloc (&d_lambda_y, cells_number*sizeof(float)));
   cutilSafeCall (hipMalloc (&d_lambda_z, cells_number*sizeof(float)));
   cutilSafeCall (hipMalloc (&d_Ek, sizeof(float)*neighbours_number*cells_number));
   cutilSafeCall (hipMalloc (&d_neighbours, sizeof(int)*neighbours_number*cells_number));

   // Set Memory
   cutilSafeCall (hipMemcpy (d_polarization, h_polarization, cells_number*sizeof(float), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_lambda_x, h_lambda_x, cells_number*sizeof(float), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_lambda_y, h_lambda_y, cells_number*sizeof(float), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_lambda_z, h_lambda_z, cells_number*sizeof(float), hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_Ek, h_Ek, sizeof(float)*neighbours_number*cells_number, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpy (d_neighbours, h_neighbours, sizeof(int)*neighbours_number*cells_number, hipMemcpyHostToDevice));

   // Set Constants
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_clock_prefactor"), &(optimization_options->clock_prefactor), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_clock_shift"), &(optimization_options->clock_shift), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_four_pi_over_number_samples"), &(optimization_options->four_pi_over_number_samples), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_two_pi_over_number_samples"), &(optimization_options->two_pi_over_number_samples), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("optimization_options_hbar_over_kBT"), &(optimization_options->hbar_over_kBT), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_clock_low"), &(options->clock_low), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_clock_high"), &(options->clock_high), sizeof(float), 0, hipMemcpyHostToDevice));  
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_clock_shift"), &(options->clock_shift), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_relaxation"), &(options->relaxation), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_time_step"), &(options->time_step), sizeof(float), 0, hipMemcpyHostToDevice));
   cutilSafeCall (hipMemcpyToSymbol(HIP_SYMBOL("options_algorithm"), &(options->algorithm), sizeof(int), 0, hipMemcpyHostToDevice));

   // For each sample...
   for (i = 0; i < iterations; i++)
   {
      printf("Inizio simulazione -- Iterazione %d\n", i); 

      // Launch Kernel
      kernelIterationParallel<<< grid, threads >>> (d_next_polarization, d_polarization, d_lambda_x, d_lambda_y, d_lambda_z, d_Ek, d_neighbours, cells_number, neighbours_number, i, 10);

      // Wait Device
      hipDeviceSynchronize ();

      cudaPrintfDisplay(stdout, true);

      // Set Memory for the next iteration
      cutilSafeCall (hipMemcpy (d_polarization, d_next_polarization, cells_number*sizeof(float), hipMemcpyDeviceToDevice));

      // Test -- Get desidered iteration results from GPU
      cutilSafeCall (hipMemcpy (h_polarization, d_next_polarization, cells_number*sizeof(float), hipMemcpyDeviceToHost));
   }

   // Free-up resources
   cudaPrintfEnd();
   hipFree(d_next_polarization);
   hipFree(d_polarization);
   hipFree(d_Ek);
   hipFree(d_neighbours);  

}



